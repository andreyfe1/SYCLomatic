// RUN: dpct --format-range=none --report-type=all -out-root %T/cusolverHelper %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverHelper/cusolverHelper.dp.cpp --match-full-lines %s

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

// CHECK: #define MACRO_A cusolverDnCreate
#define MACRO_A hipsolverDnCreate

// CHECK: void foo(int, int, int, int, int, int, int, int) {}
void foo(hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t) {}

// CHECK: void foo2(int){}
void foo2(hipsolverStatus_t){}

// CHECK: int foo3(int m, int n)
hipsolverStatus_t foo3(int m, int n)
{
    // CHECK: return 0;
    return HIPSOLVER_STATUS_SUCCESS;
}

// CHECK: extern sycl::queue* cusolverH2 = NULL;
extern hipsolverHandle_t cusolverH2 = NULL;

int main(int argc, char *argv[])
{
    // CHECK: sycl::queue* cusolverH = NULL;
    // CHECK-NEXT: int status = 0;
    // CHECK-NEXT: status = 1;
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;

    // CHECK: foo(0, 1, 2, 3, 4, 6, 7, 8);
    foo(HIPSOLVER_STATUS_SUCCESS, HIPSOLVER_STATUS_NOT_INITIALIZED, HIPSOLVER_STATUS_ALLOC_FAILED, HIPSOLVER_STATUS_INVALID_VALUE, HIPSOLVER_STATUS_ARCH_MISMATCH, HIPSOLVER_STATUS_EXECUTION_FAILED, HIPSOLVER_STATUS_INTERNAL_ERROR, HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED);

    // CHECK: cusolverH = &q_ct1;
    hipsolverDnCreate(&cusolverH);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = (cusolverH = &q_ct1, 0);
    status = hipsolverDnCreate(&cusolverH);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = (cusolverH = &q_ct1, 0);
    status = MACRO_A(&cusolverH);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = (cusolverH = nullptr, 0);
    status = hipsolverDnDestroy(cusolverH);

    // CHECK: int a = sizeof(int);
    // CHECK-NEXT: int b = sizeof(sycl::queue*);
    int a = sizeof(hipblasStatus_t);
    int b = sizeof(hipsolverHandle_t);
}


