// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/DnnlUtils/api_test18_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/DnnlUtils/api_test18_out/MainSourceFiles.yaml | wc -l > %T/DnnlUtils/api_test18_out/count.txt
// RUN: FileCheck --input-file %T/DnnlUtils/api_test18_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DnnlUtils/api_test18_out

// CHECK: 14
// TEST_FEATURE: DnnlUtils_batch_normalization_forward_inference_ex_norm
// TEST_FEATURE: DnnlUtils_batch_normalization_mode
// TEST_FEATURE: DnnlUtils_batch_normalization_ops
// TEST_FEATURE: DnnlUtils_get_batch_normalization_workspace_size

#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, scalebiasTensor;
    hipStream_t stream1;
    int in = 2, ic = 4, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 5, ow = 5;
    int sbn = 1, sbc = 4, sbh = 5, sbw = 5;
    int ele_num = in* ic * ih * iw;
    int oele_num = on* oc * oh * ow;
    int sele_num = sbn*sbc * sbh * sbw;
    int save = 1;
    float *data, *out, *scale, *bias, *rmean, *rvar, *smean, *svar, *z;
    float alpha = 1.f, beta = 0.f, eps = 1.f;
    double factor = 0.5f;

    cudnnActivationDescriptor_t ActivationDesc;
    cudnnCreateActivationDescriptor(&ActivationDesc);
    cudnnSetActivationDescriptor(ActivationDesc, CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0.0f);
    auto status = cudnnNormalizationForwardInference(
        handle, 
        CUDNN_NORM_PER_ACTIVATION,
        //CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        &alpha,
        &beta,
        dataTensor,
        data,
        scalebiasTensor,
        scale,
        bias,
        scalebiasTensor,
        smean,
        svar,
        dataTensor,
        z,
        ActivationDesc,
        outTensor,
        out,
        eps,
        1);

    return 0;
}