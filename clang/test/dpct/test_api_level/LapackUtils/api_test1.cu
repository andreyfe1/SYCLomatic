// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test1_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test1_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test1_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test1_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test1_out

// CHECK: 2
// TEST_FEATURE: LapackUtils_sygvd


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  float *a_s, *b_s, *w_s, *work_s;
  int lwork_s;
  int *devInfo;
  hipsolverDnSsygvd(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s, 3, b_s, 3, w_s, work_s, lwork_s, devInfo);
  return 0;
}
